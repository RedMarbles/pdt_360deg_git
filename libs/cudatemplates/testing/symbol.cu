#include "hip/hip_runtime.h"
#include <cudatemplates/copy.hpp>
#include <cudatemplates/devicememorylinear.hpp>
#include <cudatemplates/hostmemoryheap.hpp>
#include <cudatemplates/symbol.hpp>


#define SIZE 1024


__constant__ float cdata1[SIZE], cdata2[SIZE];
__device__ float ddata1[SIZE], ddata2[SIZE];

Cuda::Symbol<float, 1> symbol(Cuda::Size<1>(1024), cdata1);


__global__ void
kernel(float *data1, float *data2)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  data1[i] = cdata2[i];
  data2[i] = ddata2[i];
}

int
main()
{
  Cuda::Size<1> size(SIZE);
  Cuda::HostMemoryHeap<float, 1> host1(size), host2(size), host3(size), host4(size), host5(size);
  Cuda::DeviceMemoryLinear<float, 1> device1(size), device2(size), device4(size), device5(size);
  Cuda::Symbol<float, 1> csym1(size, cdata1), csym2(size, cdata2);
  Cuda::Symbol<float, 1> dsym1(size, ddata1), dsym2(size, ddata2);

  // init data:
  int seed = time(0);
  srand(seed);

  for(int i = SIZE; i--;)
    host1[i] = rand();

  // roundtrip:
  copy(csym1  , host1  );
  copy(device1, csym1  );
  copy(csym2  , device1);
  copy(host2  , csym2  );
  copy(dsym1  , host2  );
  copy(device2, dsym1  );
  copy(dsym2  , device2);
  copy(host3  , dsym2  );

#if 0
  // these produce a runtime error:
  // "cannot take the address of __constant__ data"
  float *cbuf1 = csym1.getBuffer();
  float *cbuf2 = csym2.getBuffer();
#endif

  float *dbuf1 = dsym1.getBuffer();
  float *dbuf2 = dsym2.getBuffer();

  dim3 blockDim(32);
  dim3 gridDim(SIZE / blockDim.x);
  kernel<<<gridDim, blockDim>>>(device4.getBuffer(), device5.getBuffer());
  copy(host4, device4);
  copy(host5, device5);

  // verify data:
  srand(seed);

  for(int i = SIZE; i--;) {
    int r = rand();
    assert(host3[i] == r);
    assert(host4[i] == r);
    assert(host5[i] == r);
  }
}
