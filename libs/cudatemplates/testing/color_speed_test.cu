#include "hip/hip_runtime.h"
#include <cudatemplates/cuda_gcc43_compat.hpp>

#include <iostream>

#include <cudatemplates/devicememorypitched.hpp>

#include <hip/hip_runtime.h>

#include <sys/time.h>

using namespace std;

#include "color_speed_test_kernels.cu"

/**
Does a speed test for color images
*/

double getTime()
{
  hipDeviceSynchronize();
  timeval time;
  gettimeofday(&time, NULL);
  return time.tv_sec * 1000.0 + time.tv_usec / 1000.0;
}

  inline unsigned int divUp(unsigned int a, unsigned int b) {
//     return (a + b - 1) / b;
    return (a % b != 0) ? (a / b + 1) : (a / b);
  }

int CUDAtestMemLoad(int num, int width, int height)
{
  cout << "Testing Color Image loads" << endl;
  cout << "  Number of calls = " << num << endl;
  cout << "  Size            = " << width << "x" << height << endl;

  int block_size = 16;

  Cuda::Size<2> interleaved_size(divUp(width, block_size)*block_size, divUp(height, block_size)*block_size);
  Cuda::Size<3> plane_size(divUp(width, block_size)*block_size, divUp(height, block_size)*block_size,3);
  Cuda::DeviceMemoryPitched<float4, 2> interleaved_image_in(interleaved_size);
  Cuda::DeviceMemoryPitched<float4, 2> interleaved_image_out(interleaved_size);
  Cuda::DeviceMemoryPitched<float3, 2> interleaved3_image_in(interleaved_size);
  Cuda::DeviceMemoryPitched<float3, 2> interleaved3_image_out(interleaved_size);
  Cuda::DeviceMemoryPitched<char4, 2> interleaved_image_char_in(interleaved_size);
  Cuda::DeviceMemoryPitched<char4, 2> interleaved_image_char_out(interleaved_size);
  Cuda::DeviceMemoryPitched<float, 3> plane_image_in(plane_size);
  Cuda::DeviceMemoryPitched<float, 3> plane_image_out(plane_size);
  Cuda::DeviceMemoryPitched<char, 3> plane_image_char_in(plane_size);
  Cuda::DeviceMemoryPitched<char, 3> plane_image_char_out(plane_size);
  CUDA_CHECK(hipGetLastError());

  interleaved_image_in.initMem(0);
  interleaved_image_out.initMem(0);
  interleaved3_image_in.initMem(0);
  interleaved3_image_out.initMem(0);
  interleaved_image_char_in.initMem(0);
  interleaved_image_char_out.initMem(0);
  plane_image_in.initMem(0);
  plane_image_out.initMem(0);
  plane_image_char_in.initMem(0);
  plane_image_char_out.initMem(0);
  CUDA_CHECK(hipGetLastError());

  // prepare fragmentation for processing
  dim3 dimBlock(block_size, block_size, 1);
  dim3 dimGrid(divUp(width, block_size), divUp(height, block_size), 1);

  cout << "float4 interleaved image          -  ";
  hipDeviceSynchronize();
  double start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedKernel<<<dimGrid, dimBlock>>>( interleaved_image_in.getBuffer(),
                                                      interleaved_image_out.getBuffer(),
                                                      width, height,
                                                      interleaved_image_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "float4 interleaved image (direct) -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedDirectKernel<<<dimGrid, dimBlock>>>( interleaved_image_in.getBuffer(),
                                                            interleaved_image_out.getBuffer(),
                                                            width, height,
                                                            interleaved_image_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "float3 interleaved image          -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedKernel<<<dimGrid, dimBlock>>>( interleaved3_image_in.getBuffer(),
                                                      interleaved3_image_out.getBuffer(),
                                                      width, height,
                                                      interleaved3_image_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "float3 interleaved image (direct) -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedDirectKernel<<<dimGrid, dimBlock>>>( interleaved3_image_in.getBuffer(),
                                                            interleaved3_image_out.getBuffer(),
                                                            width, height,
                                                            interleaved3_image_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "float 3-plane image               -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferPlaneKernel<<<dimGrid, dimBlock>>>( plane_image_in.getBuffer(),
                                                plane_image_out.getBuffer(),
                                                width, height,
                                                plane_image_in.stride[0],
                                                plane_image_in.stride[1]);

    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;

  cout << "char4 interleaved image           -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedKernel<<<dimGrid, dimBlock>>>( interleaved_image_char_in.getBuffer(),
                                                      interleaved_image_char_out.getBuffer(),
                                                      width, height,
                                                      interleaved_image_char_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "char4 interleaved image (direct)  -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferInterleavedDirectKernel<<<dimGrid, dimBlock>>>( interleaved_image_char_in.getBuffer(),
                                                            interleaved_image_char_out.getBuffer(),
                                                            width, height,
                                                            interleaved_image_char_in.stride[0]);
    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;
  CUDA_CHECK(hipGetLastError());

  cout << "char 3-plane image                -  ";
  hipDeviceSynchronize();
  start_time = getTime();
  for (int i=0; i<num; i++)
  {
    transferPlaneKernel<<<dimGrid, dimBlock>>>( plane_image_char_in.getBuffer(),
                                                plane_image_char_out.getBuffer(),
                                                width, height,
                                                plane_image_char_in.stride[0],
                                                plane_image_char_in.stride[1]);

    CUDA_CHECK(hipGetLastError());
    hipDeviceSynchronize();
  }
  hipDeviceSynchronize();
  cout << getTime() - start_time << endl;

  cout << endl << endl;
  return 0;
}

int
main()
{
  CUDAtestMemLoad(5000, 512, 512);
  CUDAtestMemLoad(5000, 327, 571);
  CUDAtestMemLoad(20000, 34, 23);
  CUDAtestMemLoad(1500, 1949, 1003);
  CUDAtestMemLoad(5000, 640, 480);
  CUDAtestMemLoad(10000, 320, 240);

  return 0;
}

