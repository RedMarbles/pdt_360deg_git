
#include <hip/hip_runtime.h>
template<typename U>
__global__ void transferInterleavedDirectKernel( U* input, U* output,
                                                 int width, int height, int p)
{
  // calculate absolute coordinates
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int c = y*p+x;

  // Thread index
  int tx = threadIdx.x+1;
  int ty = threadIdx.y+1;

  // Define arrays for shared memory
  __shared__ U data_shared[16+2][16+2];

  // load data into shared memory
  data_shared[ty][tx] = input[c];

  __syncthreads();

  if (x == 0)
    data_shared[ty][tx-1] = data_shared[ty][tx];
  else if (tx == 1)
    data_shared[ty][tx-1] = input[c-1];

  if (y == 0)
    data_shared[ty-1][tx] = data_shared[ty][tx];
  else if (ty == 1)
    data_shared[ty-1][tx] = input[c-p];

  if (x >= width-1)
    data_shared[ty][tx+1] = data_shared[ty][tx];
  else if (tx == 16-1)
    data_shared[ty][tx+1] = input[c+1];

  if (y >= height-1)
    data_shared[ty+1][tx] = data_shared[ty][tx];
  else if (ty == 16-1)
    data_shared[ty+1][tx] = input[c+p];

  if ((x<width) && (y<height))
  {
    output[c] = data_shared[ty][tx];
  }
}


__global__ void transferInterleavedKernel( float4* input, float4* output,
                                           int width, int height, int p)
{
  // calculate absolute coordinates
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int c = y*p+x;

  // Thread index
  int tx = threadIdx.x+1;
  int ty = threadIdx.y+1;

  // Define arrays for shared memory
  __shared__ float data_shared[16+2][16+2][3];

  // load data into shared memory
  float4 temp = input[c];
  data_shared[ty][tx][0] = temp.x;
  data_shared[ty][tx][1] = temp.y;
  data_shared[ty][tx][2] = temp.z;

  __syncthreads();

  if (x == 0)
  {
    data_shared[ty][tx-1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx-1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx-1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 1)
  {
    temp = input[c-1];
    data_shared[ty][tx-1][0] = temp.x;
    data_shared[ty][tx-1][1] = temp.y;
    data_shared[ty][tx-1][2] = temp.z;
  }

  if (y == 0)
  {
    data_shared[ty-1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty-1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty-1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 1)
  {
    temp = input[c-p];
    data_shared[ty-1][tx][0] = temp.x;
    data_shared[ty-1][tx][1] = temp.y;
    data_shared[ty-1][tx][2] = temp.z;
  }

  if (x >= width-1)
  {
    data_shared[ty][tx+1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx+1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx+1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 16-1)
  {
    temp = input[c+1];
    data_shared[ty][tx+1][0] = temp.x;
    data_shared[ty][tx+1][1] = temp.y;
    data_shared[ty][tx+1][2] = temp.z;
  }

  if (y >= height-1)
  {
    data_shared[ty+1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty+1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty+1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 16-1)
  {
    temp = input[c+p];
    data_shared[ty+1][tx][0] = temp.x;
    data_shared[ty+1][tx][1] = temp.y;
    data_shared[ty+1][tx][2] = temp.z;
  }

  if ((x<width) && (y<height))
  {
    output[c] = make_float4(data_shared[ty][tx][0], data_shared[ty][tx][1], data_shared[ty][tx][2], 1.0f);
  }
}

__global__ void transferInterleavedKernel( float3* input, float3* output,
                                           int width, int height, int p)
{
  // calculate absolute coordinates
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int c = y*p+x;

  // Thread index
  int tx = threadIdx.x+1;
  int ty = threadIdx.y+1;

  // Define arrays for shared memory
  __shared__ float data_shared[16+2][16+2][3];

  // load data into shared memory
  float3 temp = input[c];
  data_shared[ty][tx][0] = temp.x;
  data_shared[ty][tx][1] = temp.y;
  data_shared[ty][tx][2] = temp.z;

  __syncthreads();

  if (x == 0)
  {
    data_shared[ty][tx-1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx-1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx-1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 1)
  {
    temp = input[c-1];
    data_shared[ty][tx-1][0] = temp.x;
    data_shared[ty][tx-1][1] = temp.y;
    data_shared[ty][tx-1][2] = temp.z;
  }

  if (y == 0)
  {
    data_shared[ty-1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty-1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty-1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 1)
  {
    temp = input[c-p];
    data_shared[ty-1][tx][0] = temp.x;
    data_shared[ty-1][tx][1] = temp.y;
    data_shared[ty-1][tx][2] = temp.z;
  }

  if (x >= width-1)
  {
    data_shared[ty][tx+1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx+1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx+1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 16-1)
  {
    temp = input[c+1];
    data_shared[ty][tx+1][0] = temp.x;
    data_shared[ty][tx+1][1] = temp.y;
    data_shared[ty][tx+1][2] = temp.z;
  }

  if (y >= height-1)
  {
    data_shared[ty+1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty+1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty+1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 16-1)
  {
    temp = input[c+p];
    data_shared[ty+1][tx][0] = temp.x;
    data_shared[ty+1][tx][1] = temp.y;
    data_shared[ty+1][tx][2] = temp.z;
  }

  if ((x<width) && (y<height))
  {
    output[c] = make_float3(data_shared[ty][tx][0], data_shared[ty][tx][1], data_shared[ty][tx][2]);
  }
}

__global__ void transferInterleavedKernel( char4* input, char4* output,
                                           int width, int height, int p)
{
  // calculate absolute coordinates
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int c = y*p+x;

  // Thread index
  int tx = threadIdx.x+1;
  int ty = threadIdx.y+1;

  // Define arrays for shared memory
  __shared__ char data_shared[16+2][16+2][3];

  // load data into shared memory
  char4 temp = input[c];
  data_shared[ty][tx][0] = temp.x;
  data_shared[ty][tx][1] = temp.y;
  data_shared[ty][tx][2] = temp.z;

  __syncthreads();

  if (x == 0)
  {
    data_shared[ty][tx-1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx-1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx-1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 1)
  {
    temp = input[c-1];
    data_shared[ty][tx-1][0] = temp.x;
    data_shared[ty][tx-1][1] = temp.y;
    data_shared[ty][tx-1][2] = temp.z;
  }

  if (y == 0)
  {
    data_shared[ty-1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty-1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty-1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 1)
  {
    temp = input[c-p];
    data_shared[ty-1][tx][0] = temp.x;
    data_shared[ty-1][tx][1] = temp.y;
    data_shared[ty-1][tx][2] = temp.z;
  }

  if (x >= width-1)
  {
    data_shared[ty][tx+1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx+1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx+1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 16-1)
  {
    temp = input[c+1];
    data_shared[ty][tx+1][0] = temp.x;
    data_shared[ty][tx+1][1] = temp.y;
    data_shared[ty][tx+1][2] = temp.z;
  }

  if (y >= height-1)
  {
    data_shared[ty+1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty+1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty+1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 16-1)
  {
    temp = input[c+p];
    data_shared[ty+1][tx][0] = temp.x;
    data_shared[ty+1][tx][1] = temp.y;
    data_shared[ty+1][tx][2] = temp.z;
  }

  if ((x<width) && (y<height))
  {
    output[c] = make_char4(data_shared[ty][tx][0], data_shared[ty][tx][1], data_shared[ty][tx][2], 1);
  }
}

template<typename U>
__global__ void transferPlaneKernel( U* input, U* output,
                                     int width, int height, int p, int pitchY)
{
  // calculate absolute coordinates
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int c = y*p+x;

  // Thread index
  int tx = threadIdx.x+1;
  int ty = threadIdx.y+1;

  // Define arrays for shared memory
  __shared__ U data_shared[18][18][3];

  // load data into shared memory
  data_shared[ty][tx][0] = input[c];
  data_shared[ty][tx][1] = input[c+pitchY];
  data_shared[ty][tx][2] = input[c+2*pitchY];

  __syncthreads();

  if (x == 0)
  {
    data_shared[ty][tx-1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx-1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx-1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 1)
  {
    data_shared[ty][tx-1][0] = input[c-1];
    data_shared[ty][tx-1][1] = input[c-1+pitchY];
    data_shared[ty][tx-1][2] = input[c-1+2*pitchY];
  }

  if (y == 0)
  {
    data_shared[ty-1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty-1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty-1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 1)
  {
    data_shared[ty-1][tx][0] = input[c-p];
    data_shared[ty-1][tx][1] = input[c-p+pitchY];
    data_shared[ty-1][tx][2] = input[c-p+2*pitchY];
  }

  if (x >= width-1)
  {
    data_shared[ty][tx+1][0] = data_shared[ty][tx][0];
    data_shared[ty][tx+1][1] = data_shared[ty][tx][1];
    data_shared[ty][tx+1][2] = data_shared[ty][tx][2];
  }
  else if (tx == 16)
  {
    data_shared[ty][tx+1][0] = input[c+1];
    data_shared[ty][tx+1][1] = input[c+1+pitchY];
    data_shared[ty][tx+1][2] = input[c+1+2*pitchY];
  }

  if (y >= height-1)
  {
    data_shared[ty+1][tx][0] = data_shared[ty][tx][0];
    data_shared[ty+1][tx][1] = data_shared[ty][tx][1];
    data_shared[ty+1][tx][2] = data_shared[ty][tx][2];
  }
  else if (ty == 16)
  {
    data_shared[ty+1][tx][0] = input[c+p];
    data_shared[ty+1][tx][1] = input[c+p+pitchY];
    data_shared[ty+1][tx][2] = input[c+p+2*pitchY];
  }

  if ((x<width) && (y<height))
  {
    output[c] = data_shared[ty][tx][0];
    output[c+pitchY] = data_shared[ty][tx][0];
    output[c+2*pitchY] = data_shared[ty][tx][0];
  }
}


