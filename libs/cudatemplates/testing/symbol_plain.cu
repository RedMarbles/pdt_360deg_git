#include <assert.h>

#include <hip/hip_runtime.h>


#define SIZE 1024


__constant__ float data[SIZE];


int
main()
{
  size_t symsize1, symsize2;
  hipGetSymbolSize(&symsize1, HIP_SYMBOL(data));
  hipGetSymbolSize(&symsize2, HIP_SYMBOL("data"));
  assert(symsize1 == SIZE * sizeof(data[0]));
  assert(symsize2 == SIZE * sizeof(data[0]));
}
